#include "hip/hip_runtime.h"
#ifndef __HIPCC__
	#define __HIPCC__
#endif

#include <optix_world.h>
#include <optix.h>

using namespace optix;

rtDeclareVariable(float2, mousePos, , );
rtDeclareVariable(rtObject, top_object, , );


RT_PROGRAM void raytraceExecution() 
{  
	 rtPrintf("THIS. IS. CUDAAAAAAAAA! %f %f", mousePos.x, mousePos.y);
	 float3 origin = make_float3(mousePos.x, mousePos.y, 1.5);
	 float3 direction = make_float3(0,0, -1);
	 optix::Ray ray(origin, direction, 0, 0, RT_DEFAULT_MAX);
	 bool hit = false;

	 rtTrace(top_object, ray, hit);

} 