#include "hip/hip_runtime.h"
#include "parallellism.cuh"


std::vector<parallellism::Tripl> parallellism::runCalculateRadiosityMatrix(vertex::MeshS& mesh) {
	std::vector<Tripl> tripletList = {};
	int numtriangles = mesh.triangleIndices.size();

	//Load mesh into cuda memory
	glm::vec3* vertices;// = new glm::vec3[mesh.vertices.size()];
	glm::vec3* normals;// = new glm::vec3[mesh.normals.size()];
	vertex::TriangleIndex* triangleIndices;// = new vertex::TriangleIndex[mesh.triangleIndices.size()];
	hipMallocManaged(&vertices, mesh.vertices.size()*sizeof(glm::vec3));
	hipMallocManaged(&normals, mesh.normals.size()*sizeof(glm::vec3));
	hipMallocManaged(&triangleIndices, mesh.triangleIndices.size()*sizeof(vertex::TriangleIndex));
	cudaCheckError();
	std::copy(std::begin(mesh.vertices), std::end(mesh.vertices), vertices);
	std::copy(std::begin(mesh.normals), std::end(mesh.normals), normals);
	std::copy(mesh.triangleIndices.begin(), mesh.triangleIndices.end(), triangleIndices);


	int numfilled = 0;

	int rowStride = std::min(1000000/numtriangles, numtriangles);
	

	for (int row = 0; row < numtriangles; row += rowStride) {
		// calulate form factors current patch to all other patches (that have not been calculated already):
		// matrix shape should be as follows:
		// *------*------*------*
		// |   0  | 0->1 | 0->2 |
		// *------*------*------*
		// | 1->0 |  0   | 1->2 |
		// *------*------*------*
		// | 2->0 | 2->1 |   0  |
		// *------*------*------*
		//
		// such that we can do the following calculation:
		// M*V1 = V2 where
		// M is radiosity matrix
		// V1 is a vector containing the light that is emitted per patch 
		// V2 is a vector containing the light that is emitted per patch after the bounce
		Tripl* tripletRow;// = new Tripl[numtriangles];
		hipMallocManaged(&tripletRow, numtriangles*sizeof(Tripl)*rowStride);
		cudaCheckError();

		int colThreads = (256 + rowStride -1) / rowStride;
		dim3 blockDim(rowStride, colThreads);
		dim3 gridDim(1, (numtriangles + colThreads - 1)/colThreads);


		auto start = std::chrono::high_resolution_clock::now();
		calculateRow<<<gridDim, blockDim>>>(row, std::min(numtriangles, row + rowStride), tripletRow, vertices, normals, triangleIndices, numtriangles);
		hipDeviceSynchronize();
		cudaCheckError();

		auto finish = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> elapsed = finish - start;
		//std::cout << "Elapsed time: " << elapsed.count() << " s\n";

		tripletList.insert(tripletList.end(), tripletRow, &tripletRow[numtriangles*rowStride]);

		hipFree(tripletRow);

		// draw progress bar
		numfilled += numtriangles*rowStride;
		int barWidth = 70;
		float progress = float(float(numfilled) / float(numtriangles*(numtriangles - 1)));
		std::cout << "[";
		int pos = barWidth * progress;
		for (int i = 0; i < barWidth; ++i) {
			if (i < pos) std::cout << "=";
			else if (i == pos) std::cout << ">";
			else std::cout << " ";
		}

		std::cout << "] " << int(progress * 100.0) << " %\r";
		std::cout.flush();
	}

	//for (int i = 0; i < tripletList.size(); i++) {
	//	std::cout << "Form factor at index " << i << " with index (" << tripletList[i].m_row << " ,  " << tripletList[i].m_col << " is : " << tripletList[i].m_value << std::endl;
	//}
	return tripletList;
}

__global__
void parallellism::calculateRow(int rowStart, int rowEnd, Tripl* rowTripletList,
glm::vec3* vertices, glm::vec3* normals, vertex::TriangleIndex* triangleIndices, int numtriangles) {
	int threadRow = (blockIdx.x * blockDim.x + threadIdx.x) + rowStart;
	int threadCol = blockIdx.y * blockDim.y + threadIdx.y;
	int rowStride = blockDim.x * gridDim.x;
	int colStride = blockDim.y * gridDim.y;
	for (int row = threadRow; row < rowEnd; row += rowStride) {
		for (int col = threadCol; col < numtriangles; col += colStride) {
			float formfactorRC = p2pFormfactor(row, col, vertices, normals, triangleIndices);
			if (formfactorRC > 0.0) {
				// at place (x, y) we want the form factor y->x 
				// but as this is a col major matrix we store (x, y) at (y, x) -> confused yet?
				rowTripletList[(row - rowStart) * numtriangles + col] = { row, col, formfactorRC };
			}
			else {
				rowTripletList[(row - rowStart) * numtriangles + col] = { row, col, 0.0 };
			}
		}
	}
}

__device__
float parallellism::p2pFormfactor(int originPatch, int destPatch,
glm::vec3* vertices, glm::vec3* normals, vertex::TriangleIndex* triangleIndices) {
	// subdivide triangles and return in vector w/ 12 entries (4*3 coordinates)

	glm::vec3 origintriangles[4][3];
	glm::vec3 destinationtriangles[4][3];
	divideInFourTriangles(origintriangles, originPatch, vertices, normals, triangleIndices);
	divideInFourTriangles(destinationtriangles, destPatch, vertices, normals, triangleIndices);

	// init vectors
	glm::vec3 originpoints[4];
	glm::vec3 destinationpoints[4];

	glm::vec3 originNormal = avgNormal(originPatch, vertices, normals, triangleIndices);
	glm::vec3 destNormal = avgNormal(destPatch, vertices, normals, triangleIndices);

	// calculate centers of subdivided triangles
	for (int i = 0; i < 4; i++) {
		originpoints[i] = calculateCentre(origintriangles[i]);
		destinationpoints[i] = calculateCentre(destinationtriangles[i]);
	}
	

	float formfactor = 0;
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			vertex::Vertex arg1 = { originpoints[i], originNormal };
			vertex::Vertex arg2 = { destinationpoints[j], destNormal };
			float surface = calculateSurface(origintriangles[i])*calculateSurface(destinationtriangles[j]);
			formfactor = formfactor + calcPointFormfactor(arg1, arg2, surface);
		}
	}

	formfactor = formfactor / calculateSurface(originPatch, vertices, normals, triangleIndices);

	return formfactor;

}

__device__
void parallellism::divideInFourTriangles(glm::vec3 res[4][3], int triangleId, glm::vec3* vertices, glm::vec3* normals, vertex::TriangleIndex* triangleIndices) {
	glm::vec3 a = vertices[triangleIndices[triangleId].vertex.x];
	glm::vec3 b = vertices[triangleIndices[triangleId].vertex.y];
	glm::vec3 c = vertices[triangleIndices[triangleId].vertex.z];
	glm::vec3 innerA = ((b - a) / 2.0f) + a;
	glm::vec3 innerC = ((c - a) / 2.0f) + a;
	glm::vec3 innerB = ((b - c) / 2.0f) + c;

	glm::vec3 temp[4][3] = { { a, innerC, innerA },
	{ innerC, c, innerB },
	{ innerA, innerB, b },
	{ innerA, innerB, innerC } };

	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 3; j++) {
			res[i][j] = temp[i][j];
		}
	}
}

__device__
glm::vec3 parallellism::calculateCentre(int triangleId, glm::vec3* vertices, glm::vec3* normals, vertex::TriangleIndex* triangleIndices) {
	glm::vec3 centre = (vertices[triangleIndices[triangleId].vertex.x] +
		vertices[triangleIndices[triangleId].vertex.y] +
		vertices[triangleIndices[triangleId].vertex.z]);
	return glm::vec3(centre.x / 3, centre.y / 3, centre.z / 3);
}

__device__
glm::vec3 parallellism::calculateCentre(glm::vec3* triangle) {
	glm::vec3 centre = (triangle[0] + triangle[1] + triangle[2]);
	return glm::vec3(centre.x / 3, centre.y / 3, centre.z / 3);
}

__device__
glm::vec3 parallellism::avgNormal(int triangleId, glm::vec3* vertices, glm::vec3* normals, vertex::TriangleIndex* triangleIndices) {
	glm::vec3 avg = (normals[triangleIndices[triangleId].normal.x] +
		normals[triangleIndices[triangleId].normal.y] +
		normals[triangleIndices[triangleId].normal.z]);
	avg = glm::vec3(avg.x / 3, avg.y / 3, avg.z / 3);
	return glm::normalize(avg);
}

__device__
float parallellism::calcPointFormfactor(vertex::Vertex orig, vertex::Vertex dest, float surface) {
	float formfactor = 0;
	float dot1 = glm::dot(orig.normal, glm::normalize(dest.pos - orig.pos));
	float dot2 = glm::dot(dest.normal, glm::normalize(orig.pos - dest.pos));
	if (dot1 > 0 && dot2 > 0) {
		float length = glm::length(dest.pos - orig.pos);
		formfactor = ((dot1 * dot2) / (std::powf(length, 2)*HIP_PI)) * surface;
	}
	return formfactor;
}

__device__
float parallellism::calculateSurface(glm::vec3 a, glm::vec3 b, glm::vec3 c) {
	glm::vec3 ab = b - a;
	glm::vec3 ac = c - a;
	return 0.5*glm::length(glm::cross(ab, ac));
}

__device__
float parallellism::calculateSurface(glm::vec3* triangle) {
	return calculateSurface(triangle[0], triangle[1], triangle[2]);
}

__device__
float parallellism::calculateSurface(int triangleId, glm::vec3* vertices, glm::vec3* normals, vertex::TriangleIndex* triangleIndices) {
	return calculateSurface(
		vertices[triangleIndices[triangleId].vertex.x],
		vertices[triangleIndices[triangleId].vertex.y],
		vertices[triangleIndices[triangleId].vertex.z]);
}

