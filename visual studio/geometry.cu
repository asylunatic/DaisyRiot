#include "hip/hip_runtime.h"
#ifndef __HIPCC__
	#define __HIPCC__
#endif

#include <optix_world.h>
#include <optix.h>

using namespace optix;

rtDeclareVariable( optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersection(int prim_index) {
float3 ta = make_float3(0, 0, 0);
float3 tb = make_float3(1, 0, 0);
float3 tc = make_float3(0, 1, 0);
float3 normal = make_float3(0, 0, 1);
Aabb triangle = Aabb(ta, tb, tc);

//triangle plane = plane with normal 'normal' and origin 'ta'
float t;
if (dot(ray.direction, normal) != 0) t = dot((ta - ray.origin), normal)/dot(ray.direction, normal);
else t = -1;

float3 p = ray.origin + t * ray.direction; 
if (triangle.contains(p)) {
	rtPrintf("Hit!");
}
}

RT_PROGRAM void boundingbox(int prim_index, float result[6]) {
	float3 ta = make_float3(0, 0, 0);
	float3 tb = make_float3(1, 0, 0);
	float3 tc = make_float3(0, 1, 0);
	float3 normal = make_float3(0, 0, 1);
	Aabb triangle = Aabb(ta, tb, tc);
	result[0] = triangle.m_min.x;
	result[1] = triangle.m_min.y;
	result[2] = triangle.m_min.z;
	result[3] = triangle.m_max.x;
	result[4] = triangle.m_max.y;
	result[5] = triangle.m_max.z;
}